#include "hip/hip_runtime.h"
#ifndef _BCN_KERNEL_H_
#define _BCN_KERNEL_H_

#include "stdio.h"
#include "bcnrand.h"

/*	
 * Main Program, illustrates the usage of bcnrand
 
 * Call from the command line:  ./bcnrand 268435456 1 512 112 1
 * first argument is the length of sequence, then repeats (for accurate timing), block size, block count, seed (a number > 0)
 * 	
 * This function gives two examples for usage of BCN_RAND
 * The first example is for timing the alternative methods proposed in the paper [1]
 * 
 *
 *
 * The second example shows how to use BCN_RAND inline rather than writing to memory with a simple case of counting
 * the randomly generated real values under 0.9.
 *
 * The last example is how to use and time the combined generator, same process as in the second example
 *
 *
 *	This program is freeware. 
 *
 *	Please cite our work
	[1] Beliakov, G., Creighton, D., Johnstone, M. and Wilkin, T. 2013, Efficient implementation of Bailey and Borwein 
	pseudo-random number generator based on normal numbers, 
	Computer physics communications, vol. 184, no. 8, pp. 1999-2004.

	[2] G. Beliakov, M. Johnstone, D. Creighton, T. Wilkin, 2012, An efficient implementation of 
	Bailey and Borwein's algorithm for parallel random number generation on graphics processing
	units, Computing 94(5): 433-447.

	http://arxiv.org/abs/1206.1187, http://www.deakin.edu.au/~gleb/bcn_random.html

	J. Borwein and D. Bailey's work is available from:
	http://crd.lbl.gov/~dhbailey/dhbpapers/normal-pseudo.pdf
 *
 *	Copyright Gleb Beliakov, Tim Wilkin and Michael Johnstone, 2012
 */

int main(int argc, char **argv)
{
	if ( argc != 6 ) 
	{
		printf("Usage ./bcnrand <Number Elements> <iterations> <block size> <block count> <Seed> \nNow Exiting\n");
		exit(0);
	}
	unsigned int numElements = atoi(argv[1]);
	unsigned int numIterations = atoi(argv[2]);
	unsigned int numThreadsPerBlock = atoi(argv[3]);
	unsigned int numBlocks = atoi(argv[4]);
	double seed = atof(argv[5]);

	unsigned int workPerThread = numElements/numBlocks/numThreadsPerBlock;
	
	while(numBlocks*numThreadsPerBlock*workPerThread < numElements)
		++workPerThread;
	
	//calc even number of elements
	numElements = numBlocks*numThreadsPerBlock*workPerThread;
	
	printf("elemens, repeats, blokcsize, numblocks, seed: %d, %d, %d, %d, %f\n", numElements,numIterations,numThreadsPerBlock,numBlocks,seed);
	

	TimeBCNMethod(numElements, seed, numThreadsPerBlock, numBlocks, numIterations, workPerThread);
	
// Example 2: count the number of generated elements under 0.9 in parallel
	
	InlineGeneration(numElements, seed, numThreadsPerBlock, numBlocks,  workPerThread);

// Example 3: count the number of generated elements under 0.9 in parallel, using combined generator
	
	GenerationCombined(numElements, seed, numThreadsPerBlock, numBlocks,  workPerThread);	
}

#endif // #ifndef _BCN_KERNEL_H_
